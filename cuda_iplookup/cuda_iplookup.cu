#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <sys/time.h>
#include <sys/types.h>
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif
#ifdef _OPENMP
#include <omp.h>
#endif
#include "rtdsc.h"

/* Takeshi MATSUYA macchan@sfc.wide.ad.jp */

//#define DEBUG
#define	MAX_ROUTING_TABLE	(1000000)
#define	FIB_FILE		"ipv4.txt"

//#define MAX_TEST_ROUTE	(1488095)	// Giga Ethernet
#define MAX_TEST_ROUTE	(14880950)	// 10G Ethernet

TimeWatcher tw;

struct _fib {
	unsigned int ip;
	unsigned char subnet;
	unsigned char gw;
} *fib;
int fib_count;

unsigned int *fib_table_h, *fib_table_d;	// FIB table

unsigned int *routing_table_h, *routing_table_d;// routing table
int routing_count;

unsigned int *results_h, *results_d;		// IP lookup results

void load_fib()
{
	FILE *fp;
	int ip1, ip2, ip3, ip4, subnet;
	if ((fp = fopen(FIB_FILE, "r")) == NULL ) {
		fprintf(stderr, "file %s can not open.", FIB_FILE);
		exit (-1);
	}
	fib_count = 0;
	while ( fscanf(fp, "%d.%d.%d.%d/%d", &ip1, &ip2, &ip3, &ip4, &subnet) != EOF) {
		if (ip1 > 255 || subnet > 24)
			continue;
		(fib+fib_count)->ip = (ip1<<24) | (ip2<<16) | (ip3<<8) | ip4;
		(fib+fib_count)->subnet = subnet;
		(fib+fib_count)->gw = rand() & 0xff;
#ifdef DEBUG
		printf("%d.%d.%d.%d/%d\n", ip1, ip2, ip3, ip4, subnet);
#endif
		++fib_count;
	}
	printf("Routing tables=%d\n", fib_count);
	fclose(fp);
}

void mapping_fib()
{
	int i, subnet;
	unsigned int ip_start, ip_end, ip, addr;
	for (subnet=2; subnet<=24; ++subnet) {
#ifdef DEBUG
		printf("subnet=%d\n", subnet);
#endif
		for (i=0; i<fib_count; ++i) {
			if (fib[i].subnet == subnet) {
				ip_start = (fib+i)->ip & ~((unsigned int)(1<<(32-subnet))-1);
				ip_end   = (fib+i)->ip |  ((unsigned int)(1<<(32-subnet))-1);
#ifdef DEBUG
				printf("start=%08X, end=%08X\n", ip_start, ip_end);
#endif
				for ( ip=(ip_start>>8); ip<=(ip_end>>8); ++ip) {
					addr = (ip & 0xffffff);
					*(fib_table_h + addr) = (fib+i)->gw;
				}

			}
		}
	}
}

#ifdef __HIPCC__
__global__ void iplookup_gpu(unsigned int *fib_table, unsigned int *routing_table, unsigned int *results, int routing_count)
{
	int i;
	int total = gridDim.x * blockDim.x;			// number of total thrads
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	// get current thread ID

	for (i=routing_count*idx/total; i<(routing_count*(idx+1)/total); ++i)
		results[i] = fib_table[routing_table[i]>>8];
}
#endif

void iplookup_host(unsigned int *fib_table, unsigned int *routing_table, unsigned int *results, int routing_count)
{
	int i;

#ifdef _OPENMP
#pragma omp parallel for
#endif
	for (i=0; i<routing_count; ++i)
		results[i] = fib_table[routing_table[i]>>8];
#ifdef _OPENMP
#pragma barrier
#endif
}

int main(int argc, char **argv)
{
	int i, loss;

	routing_count = MAX_TEST_ROUTE;

#ifdef _OPENMP
//omp_set_num_threads(1);		// max number of threads
#pragma omp parallel
{
	printf("thread_num=%d of %d\n", omp_get_thread_num(), omp_get_num_threads());
}
#endif
#ifdef __HIPCC__
	dim3 blocks(128);
	dim3 threads(1);
#endif

	get_cpu_cycle_per_sec();

	// memory allocation and clear
	fib = (struct _fib *)malloc( sizeof(struct _fib)*MAX_ROUTING_TABLE);
	fib_table_h = (unsigned int *)malloc( sizeof(int)*16*1024*1024);
	routing_table_h = (unsigned int *)malloc( sizeof(int)*MAX_TEST_ROUTE);
	results_h = (unsigned int *)malloc( sizeof(int)*MAX_TEST_ROUTE);
	bzero( fib_table_h, sizeof(int)*16*1024*1024);
	bzero( routing_table_h, sizeof(int)*MAX_TEST_ROUTE);
#ifdef __HIPCC__
	hipMalloc((void **)&fib_table_d, sizeof(int)*16*1024*1024);
	hipMalloc((void **)&routing_table_d, sizeof(int)*MAX_TEST_ROUTE);
	hipMalloc((void **)&results_d, sizeof(int)*MAX_TEST_ROUTE);
#endif

	load_fib();
	mapping_fib();
	for (i=0; i<routing_count; ++i) {
		routing_table_h[i] = (rand() & 0xffff) << 16; //(fib+(rand() % fib_count))->ip;
	}

	start(&tw);
	end(&tw);
	loss = tw.end - tw.start;

	start(&tw);
	iplookup_host(fib_table_h, routing_table_h, results_h, routing_count);
	end(&tw);

	tw.end -= loss;
	printf("HOST\n");
	print_time_sec(&tw);

#ifdef __HIPCC__
	hipMemcpy(fib_table_d, fib_table_h, sizeof(int)*16*1024*1024, hipMemcpyHostToDevice);
	hipMemcpy(routing_table_d, routing_table_h, sizeof(int)*MAX_TEST_ROUTE, hipMemcpyHostToDevice);
	hipMemset( results_d, 0, sizeof(int)*MAX_TEST_ROUTE);

	start(&tw);
	iplookup_gpu <<< blocks, threads >>>(fib_table_d, routing_table_d, results_d, routing_count);
	hipDeviceSynchronize();
	end(&tw);

	tw.end -= loss;
	printf("GPU\n");
	print_time_sec(&tw);

	hipMemcpy(results_h, results_d, sizeof(int)*MAX_TEST_ROUTE, hipMemcpyDeviceToHost);
#endif


#ifdef	DEBUG
	for (i=0; i<routing_count; ++i)
//		if (results_h[i] == 0)
		printf( "%08X = %d\n", routing_table_h[i], results_h[i]);
#endif

	free(fib);
	free(fib_table_h);
	free(routing_table_h);
	free(results_h);

#ifdef __HIPCC__
	hipFree(fib_table_d);
	hipFree(routing_table_d);
	hipFree(results_d);
#endif

	exit(EXIT_SUCCESS);
}

